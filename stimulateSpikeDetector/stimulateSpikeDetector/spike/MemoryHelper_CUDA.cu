﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include "ProjectDefinitions.h"
using namespace std::chrono;

#ifdef USE_CUDA

extern "C" hipError_t SelectCUDA_GPU_Unit(void)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	return cudaStatus;
}

extern "C" hipError_t CheckForError(char * str)
{
	hipError_t cudaStatus;
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "%s launch failed: %s\n", str, hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching %s!\n", cudaStatus, str);
	}
	return cudaStatus;
}

extern "C" hipError_t AllocateCUDAData(float **dev_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_pointer, (length*width) * bytesInValue);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t AllocateCUDADataChar(char **dev_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_pointer, (length*width) * bytesInValue);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t AllocateCUDADataU16(uint16_t **dev_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_pointer, (length*width) * bytesInValue);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t AllocateCUDADataU32(uint32_t **dev_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)dev_pointer, (length*width) * bytesInValue);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t MemCpyCUDAData(float *dev_pointer, float *host_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pointer, host_pointer, ((length*width) * bytesInValue), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to device failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t MemCpyCUDADataU16(uint16_t *dev_pointer, uint16_t *host_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pointer, host_pointer, ((length*width) * bytesInValue), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to device failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t MemCpyCUDADataU32(uint32_t *dev_pointer, uint32_t *host_pointer, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_pointer, host_pointer, ((length*width) * bytesInValue), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to device failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t RetreiveResults(float *dev_result, float *result, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_result, (width*length) * bytesInValue, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to host failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t RetreiveResultsU32(uint32_t *dev_result, uint32_t *result, uint32_t length, uint32_t width, uint16_t bytesInValue)
{
	hipError_t cudaStatus;
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_result, (width*length) * bytesInValue, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to host failed!");
	}

	return cudaStatus;
}

extern "C" hipError_t CheckForCudaError(void)
{
	hipError_t cudaStatus;
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
		return cudaStatus;
	}

	return cudaStatus;
}

extern "C" void CleanUpCudaForSpikeDet(float *dev_kernel)
{
	hipFree(dev_kernel);
}

extern "C" void CleanUpCudaForSpikeDetU16(uint16_t *dev_kernel)
{
	hipFree(dev_kernel);
}

extern "C" void CleanUpCudaForSpikeDetU32(uint32_t *dev_kernel)
{
	hipFree(dev_kernel);
}

extern "C" void CleanUpCudaForSpikeDetChar(char *dev_kernel)
{
	hipFree(dev_kernel);
}

#endif
