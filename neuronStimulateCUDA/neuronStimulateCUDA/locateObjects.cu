#include "hip/hip_runtime.h"
/*
 * locateObjects.cu
 *
 * Contains functions to isolate and locate foreground objects in image.
 * Includes functions to compute difference between images,
 * static image thresholding to produce a binary BW image,
 * performing morphological image operations like erode and dilate on BW images
 *
 *  Created on: 26/09/2011
 *      Author: kimbjerge
 */
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, bmp utilities
#include "defs.h"
#include "BmpUtil.h"
#include "timer.h"

static unsigned int timerCUDA = 0;

__global__ void
diffImageByte( byte* diff, byte* back, byte* src, int stride)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  diff[row * stride + col] = abs(src[row * stride + col] - back[row * stride + col]);

}

__global__ void
erodeImageByte( byte* dst, byte* src, int width)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // read in input data from global memory
  // Structuring element
  byte pix01 = src[(row - 1) * width + col];
  byte pix10 = src[row * width + col - 1];
  byte pix11 = src[row * width + col];
  byte pix12 = src[row * width + col + 1];
  byte pix21 = src[(row + 1) * width + col];

  // Erode morphological operation
  float sum = pix01 + pix10 + pix11 + pix12 + pix21;
  byte pixel = 255;
  if (sum < 255.0f*5)
	  pixel = 0;

  dst[row * width + col] = pixel;

}

__global__ void
dilate3SEImageByte( byte* dst, byte* src, int width)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  byte pix01 = src[(row - 1) * width + col];
  byte pix10 = src[row * width + col - 1];
  byte pix11 = src[row * width + col];
  byte pix12 = src[row * width + col + 1];
  byte pix21 = src[(row + 1) * width + col];

  // Dilate morphological operation
  if ( (pix01 == 255) |
       (pix10 == 255) |
       (pix12 == 255) |
       (pix21 == 255) )
  {
	  dst[row * width + col] = 255;
  }
  else
  {
	  dst[row * width + col] = pix11;
  }

}

__global__ void
dilate5SEImageByte( byte* dst, byte* src, int width)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  byte pix11 = src[row * width + col];

  // Dilate morphological operation SE[5x5]
  if (
	   (src[(row - 2) * width + col - 1] == 255) |
	   (src[(row - 2) * width + col] == 255) |
	   (src[(row - 2) * width + col + 1] == 255) |
	   (src[(row - 1) * width + col - 1] == 255) |
	   (src[(row - 1) * width + col] == 255) |
	   (src[(row - 1) * width + col + 1] == 255) |
	   (src[row * width + col - 2] == 255) |
	   (src[row * width + col - 1] == 255) |
       (src[row * width + col + 1] == 255) |
       (src[row * width + col + 2] == 255) |
       (src[(row + 1) * width + col - 1] == 255) |
       (src[(row + 1) * width + col] == 255) |
       (src[(row + 1) * width + col + 1] == 255) |
       (src[(row + 2) * width + col - 1] == 255) |
       (src[(row + 2) * width + col] == 255) |
       (src[(row + 2) * width + col + 1] == 255)
       )
  {
	  dst[row * width + col] = 255;
  }
  else
  {
	  dst[row * width + col] = pix11;
  }

}

__global__ void
tresholdImageByte( byte* dst, byte* src, int strideDst, int strideSrc, byte th)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (src[row * strideSrc + col] > th)
  	dst[row * strideDst + col] = 255;
  else
  	dst[row * strideDst + col] = 0;

}

// Compute difference between 2 images
float DiffImages(byte *ImgDst, byte *ImgBack, byte *ImgSrc, ROI Size, int ISStride, int IBStride)
{
    byte  *Diff, *Back, *Src;
    size_t DiffStride, SrcStride, BackStride;

    DEBUG_MSG("[DiffImages]\n");

    // Allocation of device memory for 2D difference image
    cutilSafeCall(hipMallocPitch((void **)(&Diff), &DiffStride, Size.width * sizeof(byte), Size.height));
    DiffStride /= sizeof(byte);
    //DEBUG_MSG("DiffStride %d\n", DiffStride);

    // Allocation of memory for 2D background and source image in byte format
    cutilSafeCall(hipMallocPitch((void **)(&Back), &BackStride, Size.width * sizeof(byte), Size.height));
    BackStride /= sizeof(byte);
    //DEBUG_MSG("BackStride %d\n", BackStride);

    cutilSafeCall(hipMallocPitch((void **)(&Src), &SrcStride, Size.width * sizeof(byte), Size.height));
    SrcStride /= sizeof(byte);
    //DEBUG_MSG("SrcStride %d\n", SrcStride);

    //copy background image from host memory to device
    cutilSafeCall(hipMemcpy2D(Back, BackStride * sizeof(byte),
                               ImgBack, IBStride * sizeof(byte),
                               Size.width * sizeof(byte), Size.height,
                               hipMemcpyHostToDevice) );

    //copy source image from host memory to device
    cutilSafeCall(hipMemcpy2D(Src, SrcStride * sizeof(byte),
                               ImgSrc, ISStride * sizeof(byte),
                               Size.width * sizeof(byte), Size.height,
                               hipMemcpyHostToDevice) );

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid( ceil((float)Size.width / BLOCK_SIZE), ceil((float)Size.height / BLOCK_SIZE) );

    DEBUG_MSG("Grid (Blocks)    [%d,%d]\n", grid.x, grid.y);
    DEBUG_MSG("Threads in Block [%d,%d]\n", threads.x, threads.y);

    if (timerCUDA == 0) CreateTimer(&timerCUDA);
    RestartTimer(timerCUDA);
    diffImageByte<<< grid, threads >>>(Diff, Back, Src, SrcStride);
    StopTimer(timerCUDA);

    cutilSafeCall(hipMemcpy2D(ImgDst, IBStride * sizeof(byte),
                                Diff, DiffStride * sizeof(byte),
                                Size.width * sizeof(byte), Size.height,
                                hipMemcpyDeviceToHost) );

    //clean up memory
    cutilSafeCall(hipFree(Diff));
    cutilSafeCall(hipFree(Back));
    cutilSafeCall(hipFree(Src));

    return GetTimer(timerCUDA);
}

#define BOARDER_SIZE		4 // Additional boarder added to image for dilation, errosion and blurring
#define ADD_BOARDER(ptr, stride) (ptr + stride*BOARDER_SIZE*sizeof(byte) + BOARDER_SIZE*sizeof(byte));

// Performs thresholding and morphological operations like dilation and erode of image
float MorphObjects(byte *ImgDst, byte *ImgSrc, ROI Size, int Stride)
{
    byte *Src, *DstBW, *DstBWb, *Dst1, *Dst1b, *Dst2, *Dst2b;
    size_t DstStride, SrcStride;

    DEBUG_MSG("[MorphObjects]\n");

    // Allocation of memory for 2D source image in single precision format
    cutilSafeCall(hipMallocPitch((void **)(&Src), &SrcStride, Size.width * sizeof(byte), Size.height));
    SrcStride /= sizeof(byte);
    //DEBUG_MSG("SrcStride %d\n", SrcStride);

    //copy source image from host memory to device
    cutilSafeCall(hipMemcpy2D(Src, SrcStride * sizeof(byte),
                               ImgSrc, Stride * sizeof(byte),
                               Size.width * sizeof(byte), Size.height,
                               hipMemcpyHostToDevice) );

    // Allocation of device memory for 2D destination image in single precision format
    ROI SB = Size;
    SB.width += BOARDER_SIZE*2; // Add black boarders to allocated device image memory buffers
    SB.height += BOARDER_SIZE*2;

    cutilSafeCall(hipMallocPitch((void **)(&DstBW), &DstStride, SB.width * sizeof(byte), SB.height));
    cutilSafeCall(hipMallocPitch((void **)(&Dst1), &DstStride, SB.width * sizeof(byte), SB.height));
    cutilSafeCall(hipMallocPitch((void **)(&Dst2), &DstStride, SB.width * sizeof(byte), SB.height));

    //DEBUG_MSG("DstStride %d\n", DstStride);
    // Clear device memory for all images
    cutilSafeCall(hipMemset2D((void *)(DstBW), DstStride, 0, SB.width * sizeof(byte), SB.height));
    cutilSafeCall(hipMemset2D((void *)(Dst1), DstStride, 0, SB.width * sizeof(byte), SB.height));
    cutilSafeCall(hipMemset2D((void *)(Dst2), DstStride, 0, SB.width * sizeof(byte), SB.height));
    DstStride /= sizeof(byte);

    // Add black boarder to all images in 2D device memory
    // Needed since erode and dilate image structuring elements SE are [3x3] and [5x5]
    DstBWb = ADD_BOARDER(DstBW, DstStride);
    Dst1b = ADD_BOARDER(Dst1, DstStride);
    Dst2b = ADD_BOARDER(Dst2, DstStride);

    //setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid( ceil((float)Size.width / BLOCK_SIZE), ceil((float)Size.height / BLOCK_SIZE) );

    DEBUG_MSG("Grid (Blocks)    [%d,%d]\n", grid.x, grid.y);
    DEBUG_MSG("Threads in Block [%d,%d]\n", threads.x, threads.y);

    // start CUDA timer
    if (timerCUDA == 0) CreateTimer(&timerCUDA);
    RestartTimer(timerCUDA);

    // Generate BW image
    tresholdImageByte<<< grid, threads >>>(DstBWb, Src, DstStride, SrcStride, 25);
    cutilSafeCall(hipDeviceSynchronize());

    // Erode image with structuring element
    erodeImageByte<<< grid, threads >>>(Dst1b, DstBWb, DstStride);

    // Dilate image with structuring element
    //dilate3SEImageByte<<< grid, threads >>>(Dst2, Dst1, DstStride);
    dilate5SEImageByte<<< grid, threads >>>(Dst2b, Dst1b, DstStride);

    StopTimer(timerCUDA);

    cutilCheckMsg("Kernel execution failed");

    //copy eroded image from device memory to host memory in Src
    cutilSafeCall(hipMemcpy2D(ImgDst, Stride * sizeof(byte),
                                Dst2b, DstStride * sizeof(byte),
                                Size.width * sizeof(byte), Size.height,
                                hipMemcpyDeviceToHost) );

    //clean up memory
    cutilSafeCall(hipFree(Src));
    cutilSafeCall(hipFree(DstBW));
    cutilSafeCall(hipFree(Dst1));
    cutilSafeCall(hipFree(Dst2));

    //return time taken by the operation
    return GetTimer(timerCUDA);
}
